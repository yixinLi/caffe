#include "hip/hip_runtime.h"
// Copyright 2014 Yixin Li
// Based on relu_layer.cu

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include <algorithm>
#include "caffe/util/math_functions.hpp"
#include <cmath>

namespace caffe {

template <typename Dtype>
void SigmoidLayer<Dtype>::SetUp(
  const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  CHECK_EQ(bottom.size(), 1) << "Sigmoid Layer takes 1 blob as input.";
  CHECK_EQ(top->size(), 1) << "Sigmoid Layer takes 1 output.";
  int size = sqrt(bottom[0]->channels());
  (*top)[0]->Reshape(bottom[0]->num(), 1, size, size);
}


template <typename Dtype>
void SigmoidLayer<Dtype>::Forward_cpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->cpu_data();
  Dtype* top_data = (*top)[0]->mutable_cpu_data();
  const int count = bottom[0]->count();
  caffe_cpu_sigm(count, bottom_data, top_data);
 }


template <typename Dtype>
Dtype SigmoidLayer<Dtype>::Backward_cpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (propagate_down) {
    const Dtype* bottom_data = (*bottom)[0]->cpu_data();
    const Dtype* top_diff = top[0]->cpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_cpu_diff();
    const int count = (*bottom)[0]->count();
    for (int i = 0; i < count; ++i) {
      bottom_diff[i] = top_diff[i] / (bottom_data[i] * (1 - bottom_data[i]));
    }
  }
  return Dtype(0);
}

template <typename Dtype>
__global__ void SigmoidForward(const int n, const Dtype* in, Dtype* out) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    out[index] = 1 / (1 + exp(-in[index]));
  }
}

template <typename Dtype>
void SigmoidLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  SigmoidForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SigmoidBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    out_diff[index] = in_diff[index] / (in_data[index] * (1 - in_data[index]));
  }
}

template <typename Dtype>
Dtype SigmoidLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (propagate_down) {
    const Dtype* bottom_data = (*bottom)[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    const int count = (*bottom)[0]->count();
    SigmoidBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
  return Dtype(0);
}

INSTANTIATE_CLASS(SigmoidLayer);


}  // namespace caffe
